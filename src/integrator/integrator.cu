#include "hip/hip_runtime.h"
#include "integrator.h"

gm::Integrator::Integrator() {
  image = std::unique_ptr<RGBImage>(new RGBImage(IMAGE_WIDTH, IMAGE_HEIGHT));
  camera = std::shared_ptr<PerspectiveCamera>(new PerspectiveCamera(
      Vector3f(0.0f, 0.0f, 1.0f), Vector3f(0.0f, 0.0f, 0.0f),
      Vector3f(0.0f, 1.0f, 0.0f), IMAGE_WIDTH, IMAGE_HEIGHT, 70.0f));
}

// The entry point for the path tracing kernel. This should be called from
// the integrate function only
__global__ void pathtraceGPU(uint8_t *image, size_t width, size_t channels) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t pixelIdx = (row * width + col) * channels;

  image[pixelIdx] = 255;
  image[pixelIdx + 1] = 0;
  image[pixelIdx + 2] = 0;
}

/// Only a test function. Should be replaced later on
void gm::Integrator::pathtrace() {
  uint8_t *imageBuffer = image->getBuffer();
  for (uint32_t row = 0; row < image->getHeight(); ++row) {
    for (uint32_t col = 0; col < image->getWidth(); ++col) {
      Ray r = camera->generate_ray(row, col);
      Vector3f hitColor =
          ((r.direction + Vector3f(1.0f, 1.0f, 1.0f)) * 0.5) * 255.99;

      size_t pixelIdx = (row * image->getWidth() + col) * image->getChannels();
      imageBuffer[pixelIdx] = static_cast<uint8_t>(hitColor.x);
      imageBuffer[pixelIdx + 1] = static_cast<uint8_t>(hitColor.y);
      imageBuffer[pixelIdx + 2] = static_cast<uint8_t>(hitColor.z);
    }
  }
  image->writePNG("test.png");
}

void gm::Integrator::integrate() {
  // Allocate GPU memory for the image
  const size_t bufferSize = image->getSize();

  // Create a custom deleter since GPU memory needs to be freed after use
  auto gpuDeleter = [&](uint8_t *ptr) { hipFree(ptr); };
  std::shared_ptr<uint8_t> gpuImage(new uint8_t[bufferSize], gpuDeleter);
  hipMalloc((void **)&gpuImage, bufferSize);

  // Determine the grid and block dimensions. We need to allocate a grid of
  // blocks containing a thread per pixel. Initially the blocks will be 8x8 = 64
  // threads large
  dim3 blockDimensions(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDimensions((image->getWidth() / blockDimensions.x) + 1,
                      (image->getHeight() / blockDimensions.y) + 1);

  // Launch the path tracing kernel. This is the main entry point for
  // gamma's logic
  pathtraceGPU<<<gridDimensions, blockDimensions>>>(
      gpuImage.get(), image->getWidth(), image->getChannels());

  // Sync all of the threads before continuing
  hipDeviceSynchronize();

  // Copy result into CPU/host memory to write to a file
  hipMemcpy(image->getBuffer(), gpuImage.get(), bufferSize,
             hipMemcpyDeviceToHost);

  // Write image to disk
  image->writePNG("test.png");
}