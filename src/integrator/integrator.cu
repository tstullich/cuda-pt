#include "hip/hip_runtime.h"
#include "integrator.h"

gm::Integrator::Integrator(const std::string &filePath) {
  // Load our scene
  scene = std::unique_ptr<Scene>(new Scene(filePath));

  // Set some camera settings based on the output image
  scene->camera->setImagePlane(IMAGE_WIDTH, IMAGE_HEIGHT);

  // Build BVH
  bvh = std::unique_ptr<BVHDummy>(new BVHDummy(scene));

  // Initialize the final image
  image = std::unique_ptr<RGBImage>(new RGBImage(IMAGE_WIDTH, IMAGE_HEIGHT));
}

void gm::Integrator::pathtrace() {
  uint8_t *imageBuffer = image->getBuffer();
  size_t imageWidth = image->getWidth();
  size_t imageHeight = image->getHeight();
  uint32_t samplesPerPixel = 4;// Can be configured later

  for (uint32_t yCoord = 0; yCoord < imageHeight; ++yCoord) {
    for (uint32_t xCoord = 0; xCoord < imageWidth; ++xCoord) {

      // Create a sampler here for now. The rendering loop will need to be
      // rewritten anyways
      PCGSampler sampler(Vector2i(xCoord, yCoord), samplesPerPixel);
      // Initialize the pixel color
      Vector3f pixelColor(0.0f);
      for (uint32_t sample = 0; sample < samplesPerPixel; ++sample) {
        // Get a 2D sample for the camera rays
        Vector2f cameraSample = sampler.get2D();

        // Generate primary rays
        Ray ray = scene->camera->generate_ray(xCoord, yCoord, cameraSample);

        // Find an intersection point between the rays and the scene
        std::shared_ptr<Intersection> intersection =
            std::make_shared<Intersection>();
        bool hit = bvh->intersect(ray, intersection);
        if (hit) {
          std::cout << "Returned hit: " << intersection->name << " t: " << intersection->tHit << std::endl;
          pixelColor += (intersection->normal + 1.0f) * 0.5f;// Adjust the normal vector before shading
        }

        //if (!bvh->intersect(ray, intersection)) {
        //  // For now if we do not make any intersections with the scene
        //  // simply skip the light contributions for this sample. Later
        //  // the rendering loop can exit early here
        //  continue;
        //}

        // Compute scattering ray based on material BxDFs
        // Sample light sources to find path contribution. Skip for specular
        // materials

        // Sample BSDF for new path direction

        // Apply Russian roulette for early termination

        // Advance the sampler state for the next sample
        sampler.startNextSample();
      }

      /// Apply basic anti-aliasing by averaging the samples per-pixel
      pixelColor /= samplesPerPixel;

      /// Store the output color
      size_t pixelIdx = (yCoord * imageWidth + xCoord) * image->getChannels();
      imageBuffer[pixelIdx] = static_cast<uint8_t>(pixelColor.x * 255.99);
      imageBuffer[pixelIdx + 1] = static_cast<uint8_t>(pixelColor.y * 255.99);
      imageBuffer[pixelIdx + 2] = static_cast<uint8_t>(pixelColor.z * 255.99);
    }
  }
  image->writePNG("test.png");
}

// The code below will be used at a later time

// void gm::Integrator::integrate() {
//  // Allocate GPU memory for the image
//  const size_t bufferSize = image->getSize();
//
//  // Create a custom deleter since GPU memory needs to be freed after use
//  auto gpuDeleter = [&](uint8_t *ptr) { hipFree(ptr); };
//  std::shared_ptr<uint8_t> gpuImage(new uint8_t[bufferSize], gpuDeleter);
//  hipMalloc((void **)&gpuImage, bufferSize);
//
//  // Determine the grid and block dimensions. We need to allocate a grid of
//  // blocks containing a thread per pixel. Initially the blocks will be 8x8 =
//  64
//  // threads large
//  dim3 blockDimensions(BLOCK_SIZE, BLOCK_SIZE);
//  dim3 gridDimensions((image->getWidth() / blockDimensions.x) + 1,
//                      (image->getHeight() / blockDimensions.y) + 1);
//
//  // Launch the path tracing kernel. This is the main entry point for
//  // gamma's logic
//  pathtraceGPU<<<gridDimensions, blockDimensions>>>(
//      gpuImage.get(), image->getWidth(), image->getChannels());
//
//  // Sync all of the threads before continuing
//  hipDeviceSynchronize();
//
//  // Copy result into CPU/host memory to write to a file
//  hipMemcpy(image->getBuffer(), gpuImage.get(), bufferSize,
//             hipMemcpyDeviceToHost);
//
//  // Write image to disk
//  image->writePNG("test.png");
//}

// The entry point for the path tracing kernel. This should be called from
// the integrate function only
//__global__ void pathtraceGPU(uint8_t *image, size_t width, size_t channels) {
//  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
//  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
//  size_t pixelIdx = (row * width + col) * channels;
//
//  image[pixelIdx] = 255;
//  image[pixelIdx + 1] = 0;
//  image[pixelIdx + 2] = 0;
//}