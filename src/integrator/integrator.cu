#include "hip/hip_runtime.h"
#include "integrator.h"

gm::Integrator::Integrator() {
  image = std::unique_ptr<RGBImage>(new RGBImage(IMAGE_WIDTH, IMAGE_HEIGHT));
  // Place the camera at (0, 0, 1) looking in the -z direction
  camera = std::shared_ptr<PerspectiveCamera>(new PerspectiveCamera(
      Vector3f(0.0f, 0.0f, 1.0f), Vector3f(0.0f, 0.0f, 0.0f),
      Vector3f(0.0f, 1.0f, 0.0f), IMAGE_WIDTH, IMAGE_HEIGHT, 70.0f));
}

/// Only a test function. Should be replaced later on
void gm::Integrator::pathtrace() {
  uint8_t *imageBuffer = image->getBuffer();
  size_t imageWidth = image->getWidth();
  size_t imageHeight = image->getHeight();
  uint32_t samplesPerPixel = 4; // Can be configured later

  // Add a triangle going centered around (0, 0, -1)
  // Triangle triangle(Vector3f(-0.5f, -0.5f, 0.0f), Vector3f(0.5f, -0.5f,
  // 0.0f),
  //                  Vector3f(0.0f, 0.5f, 0.0f));

  Sphere sphere(Vector3f(0.0f, 0.0f, -1.0f), 1.0f);

  for (uint32_t yCoord = 0; yCoord < imageHeight; ++yCoord) {
    for (uint32_t xCoord = 0; xCoord < imageWidth; ++xCoord) {

      // Create a sampler here for now. The rendering loop will need to be
      // rewritten anyways
      PCGSampler sampler(Vector2i(xCoord, yCoord), samplesPerPixel);
      // Initialize the pixel color
      Vector3f pixelColor(0.0f);
      for (uint32_t sample = 0; sample < samplesPerPixel; ++sample) {
        // Get a 2D sample for the camera rays
        Vector2f cameraSample = sampler.get2D();

        if (xCoord == imageWidth / 2 && yCoord == imageHeight / 2) {
          std::cout << "Sample: " << cameraSample.x << ", " << cameraSample.y
                    << std::endl;
        }

        // Sample the primary rays
        Ray ray = camera->generate_ray(xCoord, yCoord, cameraSample);

        // Calculate an intersection with the scene
        std::unique_ptr<Intersection> intersection =
            std::make_unique<Intersection>();
        if (sphere.intersect(ray, intersection)) {
          // If we intersect the triangle set the hit color to red
          auto N = sphere.normal(intersection->surfacePoint);
          pixelColor +=
              (N + 1.0f) * 0.5f; // Adjust the normal vector before shading
        }

        // Advance the sampler state for the next sample
        sampler.startNextSample();
      }

      /// Apply basic anti-aliasing by averaging the samples per-pixel
      pixelColor /= samplesPerPixel;

      /// Store the output color
      size_t pixelIdx = (yCoord * imageWidth + xCoord) * image->getChannels();
      imageBuffer[pixelIdx] = static_cast<uint8_t>(pixelColor.x * 255.99);
      imageBuffer[pixelIdx + 1] = static_cast<uint8_t>(pixelColor.y * 255.99);
      imageBuffer[pixelIdx + 2] = static_cast<uint8_t>(pixelColor.z * 255.99);
    }
  }
  image->writePNG("test.png");
}

// The code below will be used at a later time

// void gm::Integrator::integrate() {
//  // Allocate GPU memory for the image
//  const size_t bufferSize = image->getSize();
//
//  // Create a custom deleter since GPU memory needs to be freed after use
//  auto gpuDeleter = [&](uint8_t *ptr) { hipFree(ptr); };
//  std::shared_ptr<uint8_t> gpuImage(new uint8_t[bufferSize], gpuDeleter);
//  hipMalloc((void **)&gpuImage, bufferSize);
//
//  // Determine the grid and block dimensions. We need to allocate a grid of
//  // blocks containing a thread per pixel. Initially the blocks will be 8x8 =
//  64
//  // threads large
//  dim3 blockDimensions(BLOCK_SIZE, BLOCK_SIZE);
//  dim3 gridDimensions((image->getWidth() / blockDimensions.x) + 1,
//                      (image->getHeight() / blockDimensions.y) + 1);
//
//  // Launch the path tracing kernel. This is the main entry point for
//  // gamma's logic
//  pathtraceGPU<<<gridDimensions, blockDimensions>>>(
//      gpuImage.get(), image->getWidth(), image->getChannels());
//
//  // Sync all of the threads before continuing
//  hipDeviceSynchronize();
//
//  // Copy result into CPU/host memory to write to a file
//  hipMemcpy(image->getBuffer(), gpuImage.get(), bufferSize,
//             hipMemcpyDeviceToHost);
//
//  // Write image to disk
//  image->writePNG("test.png");
//}

// The entry point for the path tracing kernel. This should be called from
// the integrate function only
//__global__ void pathtraceGPU(uint8_t *image, size_t width, size_t channels) {
//  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
//  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
//  size_t pixelIdx = (row * width + col) * channels;
//
//  image[pixelIdx] = 255;
//  image[pixelIdx + 1] = 0;
//  image[pixelIdx + 2] = 0;
//}