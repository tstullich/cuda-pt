#include "hip/hip_runtime.h"
#include "integrator.h"

gm::Integrator::Integrator() {
  image = std::unique_ptr<RGBImage>(new RGBImage(IMAGE_WIDTH, IMAGE_HEIGHT));
  camera = std::shared_ptr<PerspectiveCamera>(new PerspectiveCamera(
      Vector3f(0.0f, 0.0f, 1.0f), Vector3f(0.0f, 0.0f, 0.0f),
      Vector3f(0.0f, 1.0f, 0.0f), IMAGE_WIDTH, IMAGE_HEIGHT, 70.0f));
}

// The entry point for the path tracing kernel. This should be called from
// the integrate function only
__global__ void pathtraceGPU(uint8_t *image, size_t width, size_t channels) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t pixelIdx = (row * width + col) * channels;

  image[pixelIdx] = 255;
  image[pixelIdx + 1] = 0;
  image[pixelIdx + 2] = 0;
}

/// Only a test function. Should be replaced later on
void gm::Integrator::pathtrace() {
  uint8_t *imageBuffer = image->getBuffer();
  size_t imageWidth = image->getWidth();
  size_t imageHeight = image->getHeight();
  for (uint32_t yCoord = 0; yCoord < imageHeight; ++yCoord) {
    for (uint32_t xCoord = 0; xCoord < imageWidth; ++xCoord) {
      Ray r = camera->generate_ray(xCoord, yCoord);
      Vector3f hitColor = (r.direction + Vector3f(1.0f, 1.0f, 1.0f)) * 0.5;

      size_t pixelIdx = (yCoord * imageWidth + xCoord) * image->getChannels();
      imageBuffer[pixelIdx] = static_cast<uint8_t>(hitColor.x * 255.99);
      imageBuffer[pixelIdx + 1] = static_cast<uint8_t>(hitColor.y * 255.99);
      imageBuffer[pixelIdx + 2] = static_cast<uint8_t>(hitColor.z * 255.99);
    }
  }
  image->writePNG("test.png");
}

void gm::Integrator::integrate() {
  // Allocate GPU memory for the image
  const size_t bufferSize = image->getSize();

  // Create a custom deleter since GPU memory needs to be freed after use
  auto gpuDeleter = [&](uint8_t *ptr) { hipFree(ptr); };
  std::shared_ptr<uint8_t> gpuImage(new uint8_t[bufferSize], gpuDeleter);
  hipMalloc((void **)&gpuImage, bufferSize);

  // Determine the grid and block dimensions. We need to allocate a grid of
  // blocks containing a thread per pixel. Initially the blocks will be 8x8 = 64
  // threads large
  dim3 blockDimensions(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDimensions((image->getWidth() / blockDimensions.x) + 1,
                      (image->getHeight() / blockDimensions.y) + 1);

  // Launch the path tracing kernel. This is the main entry point for
  // gamma's logic
  pathtraceGPU<<<gridDimensions, blockDimensions>>>(
      gpuImage.get(), image->getWidth(), image->getChannels());

  // Sync all of the threads before continuing
  hipDeviceSynchronize();

  // Copy result into CPU/host memory to write to a file
  hipMemcpy(image->getBuffer(), gpuImage.get(), bufferSize,
             hipMemcpyDeviceToHost);

  // Write image to disk
  image->writePNG("test.png");
}