#include "hip/hip_runtime.h"
#include "camera.h"

/// TODO Add focalLength for defocus blur
gm::PerspectiveCamera::PerspectiveCamera(const Vector3f &position,
                                         const Vector3f &lookAt,
                                         const Vector3f &up, size_t imageWidth,
                                         size_t imageHeight, float fov)
    : imageWidth(imageWidth), imageHeight(imageHeight) {
  aspectRatio = static_cast<float>(imageWidth) / imageHeight;
  scale = tan((fov * 0.5f) * M_PI / 180.0f);

  // Create the camera to world matrix to transform rays
  setCameraToWorld(position, lookAt, up);
}

gm::Ray gm::PerspectiveCamera::generate_ray(uint32_t xCoord, uint32_t yCoord) {
  Vector3f origin;
  // Transform origin point using the camera-to-world matrix
  origin = cameraToWorld.multiplyPoint(origin);

  // Create a projection point on the NDC plane
  float x = (2.0f * (xCoord + 0.5f) / static_cast<float>(imageWidth) - 1.0f) *
            aspectRatio * scale;
  float y =
      (1.0f - 2.0f * (yCoord + 0.5f) / static_cast<float>(imageHeight)) * scale;

  // Position vector at the NDC plane looking in the negative z direction
  Vector3f direction(x, y, -1.0f);

  // Transform direction vector using the camera-to-world matrix and
  // normalize
  direction = normalize(cameraToWorld.multiplyVector(direction));

  return Ray(origin, direction);
}

void gm::PerspectiveCamera::setCameraToWorld(const Vector3f &position,
                                             const Vector3f &lookAt,
                                             const Vector3f &up) {
  // Create the three basis vectors for the camera orientation
  Vector3f view = normalize(position - lookAt);
  Vector3f right = normalize(cross(normalize(up), view));
  Vector3f newUp = cross(view, right);

  if (right.length() == 0.0f) {
    // Error here because the up vector and viewing direction are the same!.
    // Need to handle error here
  }

  // Set the components of the camera to world transformation matrix
  cameraToWorld[0][0] = right.x;
  cameraToWorld[0][1] = right.y;
  cameraToWorld[0][2] = right.z;
  cameraToWorld[1][0] = newUp.x;
  cameraToWorld[1][1] = newUp.y;
  cameraToWorld[1][2] = newUp.z;
  cameraToWorld[2][0] = view.x;
  cameraToWorld[2][1] = view.y;
  cameraToWorld[2][2] = view.z;
  cameraToWorld[3][0] = position.x;
  cameraToWorld[3][1] = position.y;
  cameraToWorld[3][2] = position.z;
  cameraToWorld[3][3] = 1.0f;
}