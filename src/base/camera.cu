#include "hip/hip_runtime.h"
#include "camera.h"

gm::PerspectiveCamera::PerspectiveCamera(const float &fov) {
  scale = tanf((fov * 0.5f) * M_PI / 180.0f);

  //cameraToWorld[0][0]
}

gm::PerspectiveCamera::PerspectiveCamera(const Matrix4x4f &cameraToWorld,
                                         const float &fov) : cameraToWorld(cameraToWorld) {
  // Set the scaling factor based on the fov
  scale = tanf((fov * 0.5f) * M_PI / 180.0f);
}

gm::Ray gm::PerspectiveCamera::generate_ray(uint32_t xCoord, uint32_t yCoord,
                                            const Vector2f &sample) {
  // Transform origin point using the camera-to-world matrix
  Vector3f origin = cameraToWorld.multiplyPoint(Vector3f(0.0f));

  // Create a projection point on the image plane using normalized device
  // coordinates. Move the initial point from the center using two samples
  float x =
      (2.0f * (xCoord + sample.x + 0.5f) / static_cast<float>(imageWidth) - 1.0f) * aspectRatio * scale;
  float y = (1.0f - 2.0f * (yCoord + sample.y + 0.5f) / static_cast<float>(imageHeight)) * scale;

  // Position vector at the image plane looking in the negative z direction
  Vector3f direction(x, y, -1.0f);

  // Transform direction vector using the camera-to-world matrix and
  // normalize
  direction = normalize(cameraToWorld.multiplyVector(direction));

  return {origin, direction};
}

void gm::PerspectiveCamera::setImagePlane(const size_t &width,
                                          const size_t &height) {
  imageWidth = width;
  imageHeight = height;
  aspectRatio = static_cast<float>(imageWidth) / imageHeight;
}