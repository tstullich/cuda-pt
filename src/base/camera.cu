#include "hip/hip_runtime.h"
#include "camera.h"

/// TODO Add focalLength for defocus blur
gm::PerspectiveCamera::PerspectiveCamera(const Vector3f &position,
                                         const Vector3f &lookAt,
                                         const Vector3f &up, size_t imageWidth,
                                         size_t imageHeight, float fov)
    : imageWidth(imageWidth), imageHeight(imageHeight) {
  aspectRatio = static_cast<float>(imageWidth) / imageHeight;
  scale = tan((fov * 0.5f) * M_PI / 180.0f);

  // Create the camera to world matrix to transform rays
  setCameraToWorld(position, lookAt, up);
}

gm::PerspectiveCamera::PerspectiveCamera(const Vector3f &location,
                                         const Quaternionf &rotation,
                                         const float &fov,
                                         const std::string &name)
    : SceneObject(location, rotation, Vector3f(1), name) {
  scale = tan((fov * 0.5f) * M_PI / 180.0f);
  auto rotationMatrix = rotation.toMat4();

  cameraToWorld[0][0] = rotationMatrix[0][0];
  cameraToWorld[0][1] = rotationMatrix[0][1];
  cameraToWorld[0][2] = rotationMatrix[0][2];
  cameraToWorld[1][0] = rotationMatrix[1][0];
  cameraToWorld[1][1] = rotationMatrix[1][1];
  cameraToWorld[1][2] = rotationMatrix[1][2];
  cameraToWorld[2][0] = rotationMatrix[2][0];
  cameraToWorld[2][1] = rotationMatrix[2][1];
  cameraToWorld[2][2] = rotationMatrix[2][2];
  cameraToWorld[3][0] = location.x;
  cameraToWorld[3][1] = location.y;
  cameraToWorld[3][2] = location.z;
  cameraToWorld[3][3] = 1.0f;
}


gm::Ray gm::PerspectiveCamera::generate_ray(uint32_t xCoord, uint32_t yCoord,
                                            const Vector2f &sample) {
  Vector3f origin;
  // Transform origin point using the camera-to-world matrix
  origin = cameraToWorld.multiplyPoint(origin);

  // Create a projection point on the image plane using normalized device
  // coordinates. Move the initial point from the center using two samples
  float x =
      (2.0f * (xCoord + sample.x + 0.5f) / static_cast<float>(imageWidth) -
       1.0f) *
      aspectRatio * scale;
  float y = (1.0f - 2.0f * (yCoord + sample.y + 0.5f) /
                        static_cast<float>(imageHeight)) *
            scale;

  // Position vector at the image plane looking in the negative z direction
  Vector3f direction(x, y, -1.0f);

  // Transform direction vector using the camera-to-world matrix and
  // normalize
  direction = normalize(cameraToWorld.multiplyVector(direction));

  return Ray(origin, direction);
}

void gm::PerspectiveCamera::setCameraToWorld(const Vector3f &position,
                                             const Vector3f &lookAt,
                                             const Vector3f &up) {
  // Create the three basis vectors for the camera orientation
  Vector3f view = normalize(position - lookAt);
  Vector3f right = normalize(cross(normalize(up), view));
  Vector3f newUp = cross(view, right);

  if (right.length() == 0.0f) {
    // Error here because the up vector and viewing direction are the same!.
    // Need to handle error here
  }

  // Set the components of the camera to world transformation matrix
  cameraToWorld[0][0] = right.x;
  cameraToWorld[0][1] = right.y;
  cameraToWorld[0][2] = right.z;
  cameraToWorld[1][0] = newUp.x;
  cameraToWorld[1][1] = newUp.y;
  cameraToWorld[1][2] = newUp.z;
  cameraToWorld[2][0] = view.x;
  cameraToWorld[2][1] = view.y;
  cameraToWorld[2][2] = view.z;
  cameraToWorld[3][0] = position.x;
  cameraToWorld[3][1] = position.y;
  cameraToWorld[3][2] = position.z;
  cameraToWorld[3][3] = 1.0f;
}

void gm::PerspectiveCamera::setImagePlane(const size_t &width,
                                          const size_t &height) {
  imageWidth = width;
  imageHeight = height;
  aspectRatio = static_cast<float>(imageWidth) / imageHeight;
}