#include "hip/hip_runtime.h"
#include "base/image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

gm::RGBImage::RGBImage(size_t width, size_t height)
    : width(width), height(height) {
  buffer_size = width * height * CHANNELS * sizeof(uint8_t);
  image = new uint8_t[buffer_size];
}

gm::RGBImage::~RGBImage() { free(image); }

__global__ void color(uint8_t *image, size_t width, size_t channels) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t pixel_idx = (row * width + col) * channels;

  uint8_t r = 255;
  uint8_t g = 0;
  uint8_t b = 0;
  image[pixel_idx] = r;
  image[pixel_idx + 1] = g;
  image[pixel_idx + 2] = b;
}

void gm::RGBImage::set_colors() {
  uint8_t *gpu_image;
  hipMalloc((void **)&gpu_image, buffer_size);

  dim3 block_dimensions(8, 8);
  dim3 grid_dimensions((width / block_dimensions.x) + 1,
                       (height / block_dimensions.y) + 1);

  color<<<grid_dimensions, block_dimensions>>>(gpu_image, width, CHANNELS);

  hipDeviceSynchronize();

  hipMemcpy(image, gpu_image, buffer_size, hipMemcpyDeviceToHost);

  hipFree(gpu_image);
}

void gm::RGBImage::write_png(const std::string &file_name) {
  stbi_write_png(file_name.data(), width, height, CHANNELS, image, 0);
}